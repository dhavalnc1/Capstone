// #include "../common/book.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
static void HandleError(hipError_t err, const char *file, int line)
{if (err != hipSuccess)
{printf("%s in %s at line %d\n", hipGetErrorString(err),file, line);
exit(EXIT_FAILURE);
}
}

__global__ void add( int *a, int *b, int *c ) {
 int tid = blockIdx.x; // handle the data at this index
 if (tid < N)
 c[tid] = a[tid] + b[tid];
}

int main( void ) {
 int a[N], b[N], c[N];
 int *dev_a, *dev_b, *dev_c;
 // allocate the memory on the GPU
  hipMalloc( (void**)&dev_a, N * sizeof(int) ) ;
  hipMalloc( (void**)&dev_b, N * sizeof(int) ) ;
  hipMalloc( (void**)&dev_c, N * sizeof(int) ) ;
 // fill the arrays 'a' and 'b' on the CPU
 for (int i=0; i<N; i++) {
 a[i] = -i;
 b[i] = i * i;
 }

 // copy the arrays 'a' and 'b' to the GPU
  hipMemcpy( dev_a, a, N * sizeof(int),
 hipMemcpyHostToDevice ) ;
  hipMemcpy( dev_b, b, N * sizeof(int),
 hipMemcpyHostToDevice ) ;
 add<<<N,1>>>( dev_a, dev_b, dev_c );
 // copy the array 'c' back from the GPU to the CPU
  hipMemcpy( c, dev_c, N * sizeof(int),
 hipMemcpyDeviceToHost ) ;
 // display the results
 for (int i=0; i<N; i++) {
 printf( "%d + %d = %d\n", a[i], b[i], c[i] );
 }
 // free the memory allocated on the GPU
 hipFree( dev_a );
 hipFree( dev_b );
 hipFree( dev_c );
 return 0;
}
